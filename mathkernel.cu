#include "hip/hip_runtime.h"
#include "mathkernel.cuh"

__global__ void memset(float *a, int n, float val)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < n){
        a[tid] = val;
    }
}

__global__ void vecAdd(float *a, float *b, float *c, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n){
        c[i] = a[i] + b[i];
    }
}

__global__ void matMulvec(float *a, float *b, float *c, int row, int col, bool transpose)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    float sum = 0;
    if(transpose){
        if(i < col){
            for(int j = 0; j < row; j++){
                sum += a[j * col + i] * b[j];
            }
            c[i] = sum;
        }
    }else{
        if(i < row){
            for(int j = 0; j < col; j++){
                sum += a[i * col + j] * b[j];
            }
            c[i] = sum;
        }
    }
}

__global__ void sigmoid_ztoa(float *z, float *a, int n)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < n){
        a[tid] = 1.0 / (1.0 + exp(-z[tid]));
    }
}

__global__ void sigmoid_z_prime(float *a, float *z_prime, int n)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < n){
        z_prime[tid] = a[tid] * (1 - a[tid]);
    }
}

__global__ void cost_prime(float *a, float *y, float *da, int n)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < n){
        da[tid] = a[tid] - y[tid];
    }
}

__global__ void vecMul(float *a, float *b, float *c, int n)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < n){
        c[tid] = a[tid] * b[tid];
    }
}


__global__ void copy(float *dst, float *src, int n)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < n){
        dst[tid] = src[tid];
    }
}

__global__ void update(float *v, float *dC_dv, float eta, int n)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < n){
        v[tid] -= eta * dC_dv[tid];
    }
}

__global__ void cal_dw(float *a, float *delta, float *dC_dw, int input_size, int output_size)
{
    // dC_dw(j, k) = a[l-1](k) * delta[j]
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < input_size * output_size){
        dC_dw[tid] = delta[tid / input_size] * a[tid % input_size];
    }
}

__global__ void cal_loss(float *a, float *y, float *loss, int n)
{
    extern __shared__ float sdata[];

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int sidx = threadIdx.x;

    float diff = (a[tid] - y[tid]);
    sdata[sidx] = 0.5 * diff * diff;

    __syncthreads();

    // Reduce within the block using shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (sidx < s) {
            sdata[sidx] += sdata[sidx + s];
        }
        __syncthreads();
    }

    // Atomic add to accumulate the block sums
    if (sidx == 0) {
        atomicAdd(loss, sdata[0]);
    }
}

