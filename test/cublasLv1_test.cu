#include "hip/hip_runtime.h"

#include <hipblas.h>

#include <cstdlib>
#include <cstdio>
#include <vector>

#include <hip/hip_runtime.h>
#include "../common.h"

#include <gtest/gtest.h>

using data_type = double;
hipblasHandle_t cublasH = NULL;
hipStream_t stream = NULL;

/* cublas level 1 function */
/* cublasDamax */
TEST(cublas, amax){

    /**
     * A = [1.0 2.0 3.0; 4.0]
    */
    const std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};
    const int incx = 1;

    int result = 0.0;
    data_type *d_A = NULL;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc((void**)&d_A, sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));

    CUBLAS_CHECK(hipblasIdamax(cublasH, A.size(), d_A, incx, &result));
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

    EXPECT_EQ(result, 4);
}

/* cublasDamin */
TEST(cublas, amin){
    const std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};

    const int incx =1;

    int result = 5.0;
    data_type *d_A = NULL;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));
    
    CUBLAS_CHECK(hipblasIdamin(cublasH, A.size(), d_A, incx, &result));
    CUDA_CHECK(hipStreamSynchronize(stream));

    EXPECT_EQ(result, 1);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
}

/* sum : hipblasDasum */
TEST(cublas, asum){
    const std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};

    const int incx =1;

    data_type *d_A = NULL;
    double result = 0;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));
    
    CUBLAS_CHECK(hipblasDasum(cublasH, A.size(), d_A, incx, &result));
    CUDA_CHECK(hipStreamSynchronize(stream));

    EXPECT_TRUE(result - 10 < 1e-6);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
}

/* vecMul hipblasDaxpy */
TEST(cublas, axpy){
    const std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};
    std::vector<data_type> B = {1.0, 2.0, 3.0, 4.0};
    
    const int incx =1;
    const int incy =1;

    data_type alpha = 2.1;

    data_type *d_A = NULL;
    data_type *d_B = NULL;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream));
    
    CUBLAS_CHECK(hipblasDaxpy(cublasH, A.size(), &alpha, d_A, incx, d_B, incy));
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipMemcpyAsync(B.data(), d_B, sizeof(data_type) * B.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));


    for(int i = 0; i < B.size(); ++i){
        EXPECT_TRUE(B[i] - (A[i] + alpha * A[i]) < 1e-6);
    }


    for(int i = 0; i < B.size(); ++i){
        EXPECT_TRUE(B[i] - (A[i] + alpha * A[i]) < 1e-6);
    }

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
}

/* copy : hipblasDcopy */
TEST(cublas, copy){
    const std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};
    std::vector<data_type> B(A.size(), 0);
    
    const int incx = 1;
    const int incy = 1;

    data_type alpha = 2.1;

    data_type *d_A = NULL;
    data_type *d_B = NULL;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream));
    
    CUBLAS_CHECK(hipblasDcopy(cublasH, A.size(), d_A, incx, d_B, incy));
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipMemcpyAsync(B.data(), d_B, sizeof(data_type) * B.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));


    for(int i = 0; i < B.size(); ++i){
        EXPECT_EQ(B[i], A[i]);
    }


    for(int i = 0; i < B.size(); ++i){
        EXPECT_TRUE(B[i] - (A[i] + alpha * A[i]) < 1e-6);
    }

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
}

/* also have complex number version: cublasDdotc */
TEST(cublas, dot){
    const std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};
    const std::vector<data_type> B = {5.0, 6.0, 7.0, 8.0};

    const int incx = 1;
    const int incy = 1;

    data_type result = 0;

    data_type *d_A = NULL;
    data_type *d_B = NULL;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream));
    
    CUBLAS_CHECK(hipblasDdot(cublasH, A.size(), d_A, incx, d_B, incy, &result));
    CUDA_CHECK(hipStreamSynchronize(stream));

    data_type res = 0;
    for(int i = 0; i < A.size(); i++){
        res += A[i] * B[i];
    }

    EXPECT_EQ(result, res);
    
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
}
    
TEST(cublas, nrm2){
    const std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};

    const int incx = 1;

    data_type result = 0;
    data_type *d_A = NULL;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));

    CUBLAS_CHECK(hipblasDnrm2(cublasH, A.size(), d_A, incx, &result)); 
    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
        nrm2(A) = sqrt(1*1 + 2*2 + 3*3 + 4*4)
    */
    EXPECT_TRUE(result - 5.477225575 < 1e-6);
    
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
}


TEST(cublas, rot){
    std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};
    std::vector<data_type> B = {5.0, 6.0, 7.0, 8.0};

    const int incx = 1;
    const int incy = 1;

    const data_type c = 2.1;
    const data_type s = 1.2;

    data_type *d_A = NULL;
    data_type *d_B = NULL;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream));
    
    CUBLAS_CHECK(hipblasDrot(cublasH, A.size(), d_A, incx, d_B, incy, &c, &s));

    CUDA_CHECK(hipMemcpyAsync(A.data(), d_A, sizeof(data_type)*A.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(B.data(), d_B, sizeof(data_type)*B.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    // [ A  B ] * rot(sin a = s, cos a = c) = [c*x + s*y -s*x + c* y]
    for(int i = 0; i < A.size(); i++){
        std::cout << A[i] << ' ';
    }
    std::cout << std::endl;

    for(int i = 0; i < B.size(); i++){
        std::cout << B[i] << ' ';
    }
    std::cout << std::endl;

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
}

/* https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasDrotg#cublas-t-rotg */
TEST(cublas, rotg){
    data_type A = 2.1;
    data_type B = 1.2;
    data_type c = 2.1;
    data_type s = 1.2;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUBLAS_CHECK(hipblasDrotg(cublasH, &A, &B, &c, &s));
    CUDA_CHECK(hipStreamSynchronize(stream));
    
    std::cout << A << ' ' << B << std::endl;

    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
}


TEST(cublas, rotm){
    std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};
    std::vector<data_type> B = {5.0, 6.0, 7.0, 8.0};
    std::vector<data_type> param = {1.0, 5.0, 6.0, 7.0, 8.0};

    const int incx = 1;
    const int incy = 1;

    data_type *d_A = NULL;
    data_type *d_B = NULL;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream));
    
    CUBLAS_CHECK(hipblasDrotm(cublasH, A.size(), d_A, incx, d_B, incy, param.data()));

    CUDA_CHECK(hipMemcpyAsync(A.data(), d_A, sizeof(data_type)*A.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(B.data(), d_B, sizeof(data_type)*B.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    
    for(int i = 0; i < A.size(); i++){
        std::cout << A[i] << ' ';
    }
    std::cout << std::endl;

    for(int i = 0; i < B.size(); i++){
        std::cout << B[i] << ' ';
    }
    std::cout << std::endl;

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
}


TEST(cublas, rotmg){
    data_type A = 1.0;
    data_type B = 5.0;
    data_type X = 2.1;
    data_type Y = 1.2;
    std::vector<data_type> param = {1.0, 5.0, 6.0, 7.0, 8.0};

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUBLAS_CHECK(hipblasDrotmg(cublasH, &A, &B, &X, &Y, param.data()));
    CUDA_CHECK(hipStreamSynchronize(stream));
    
    std::cout << A << ' ' << B << ' ' << X << std::endl;

    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
}


TEST(cublas, scal){

    /**
     * A = [1.0 2.0 3.0; 4.0]
    */
    const std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};
    const int incx = 1;

    const data_type alpha = 2.2;
    data_type *d_A = NULL;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc((void**)&d_A, sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));

    CUBLAS_CHECK(hipblasDscal(cublasH, A.size(), &alpha, d_A, incx));
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    for(int i = 0; i < A.size(); i++){
        std::cout << A[i] << ' ';
    }
    std::cout << std::endl;

    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

}

TEST(cublas, swap){
    std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};
    std::vector<data_type> B = {5.0, 6.0, 7.0, 8.0};

    const int incx = 1;
    const int incy = 1;

    data_type *d_A = NULL;
    data_type *d_B = NULL;

    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream));
    
    CUBLAS_CHECK(hipblasDswap(cublasH, A.size(), d_A, incx, d_B, incy));

    CUDA_CHECK(hipMemcpyAsync(A.data(), d_A, sizeof(data_type)*A.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(B.data(), d_B, sizeof(data_type)*B.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    for(int i = 0; i < A.size(); i++){
        std::cout << A[i] << ' ';
    }
    std::cout << std::endl;

    for(int i = 0; i < B.size(); i++){
        std::cout << B[i] << ' ';
    }
    std::cout << std::endl;

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
}